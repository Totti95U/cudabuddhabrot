#include "hip/hip_runtime.h"
/*
To delete "warning C4819"
1. Open property of buddhabrot project.
2. Open [CUDA C/C++]/[Command Line].
3. Write "-Xcompiler -wd4819" in additional options.
*/


#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define WIDTH 1280
#define HEIGHT 720
#define RTGRIDNUM 2048

typedef struct {
	float real;
	float imag;
} complex;

typedef struct {
	int w;
	int h;
	double ratio;

	double dx;
	double dy;

	complex center;
	double size;
	double max_real;
	double min_real;
	double max_imag;
	double min_imag;
} graphic;

typedef struct {
	int samples_per_thread;
	int min_iteration;
	int max_iteration;
} iterationContorol;

clock_t start_t, subend_t;

graphic g;
iterationContorol iteration;

hipError_t renderImage(unsigned long long int* buddha, const graphic graph, const iterationContorol iteration);

__device__ complex f(complex z, complex c) {
	complex toReturn;
	toReturn.real = z.real * z.real - z.imag * z.imag + c.real;
	toReturn.imag = 2 * z.real * z.imag + c.imag;
	return toReturn;
}

__global__ void initRNG(const unsigned int seed, hiprandStateMRG32k3a_t* states) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	hiprand_init(seed, index, 0, states + index);
}

__device__ int checkinWindow(complex z, graphic graph) {
	if (graph.min_real < z.real && z.real < graph.max_real &&
		graph.min_imag < z.imag && z.imag < graph.max_imag) {
		return 1;
	}
	return 0;
}

__device__ int checkinMainBulb(complex z) {
	float q = (z.real - 1.0 / 4.0) * (z.real - 1.0 / 4.0) + z.imag * z.imag;
	if (q * (q + (z.real - 1.0 / 4.0)) < (z.imag * z.imag) / 4.0) {
		return 1;
	}
	else {
		return 0;
	}
}

__device__ int checkinSecondDisc(complex z) {
	if ((z.real + 1) * (z.real + 1) + z.imag * z.imag < 0.25 * 0.25) {
		return 1;
	}
	else {
		return 0;
	}
}

__global__ void estImportance(int* importance, graphic graph, iterationContorol iteration) {
	int indexx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int indexy = (blockIdx.y * blockDim.y) + threadIdx.y;
	complex c, z;

	// Initiarize complex num c , z and int importance.
	c.real = -3.2 + 6.4 * indexx / RTGRIDNUM;
	c.imag = -3.2 + 6.4 * indexy / RTGRIDNUM;
	z.real = 0.0; z.imag = 0.0;
	importance[indexx + indexy * RTGRIDNUM] = 0;

	if (checkinMainBulb(c) || checkinSecondDisc(c)) {
	 	importance[indexx + indexy * RTGRIDNUM] = 0;
		return;
	}

	for (int i = 0; i < iteration.max_iteration; i++) {
		z = f(z, c);
		if (z.real * z.real + z.imag * z.imag > 10.0) {
			return;
		}
		else if (i == iteration.max_iteration - 1) {
			importance[indexx + indexy * RTGRIDNUM] = 0;
			return;
		}
		else if (checkinWindow(z, graph) && i >= iteration.min_iteration) {
			importance[indexx + indexy * RTGRIDNUM] = 1;
		}
	}
}

__device__ void draw_point(unsigned long long int* buddha, complex z, const graphic g) {
	int xnum, ynum;
	if (checkinWindow(z, g)) {
		xnum = (z.real - g.min_real) / g.dx;
		ynum = g.h - (z.imag - g.min_imag) / g.dy;

		buddha[xnum + ynum * g.w] += 1;
	}
}

__device__ complex curand_withtable(hiprandStateMRG32k3a_t* state, const complex* randTable, const int length) {
	complex toReturn;
	const int index = blockDim.x * blockIdx.x + threadIdx.x;

	int t_index = hiprand(&state[index]) % length;
	toReturn = randTable[t_index];
	toReturn.real += (-3.2 + 6.4 * hiprand_uniform(&state[index])) / RTGRIDNUM;
	toReturn.imag += (-3.2 + 6.4 * hiprand_uniform(&state[index])) / RTGRIDNUM;
	return toReturn;
}

__global__ void computeBuddhabrot(unsigned long long int* buddha, const graphic graph, const iterationContorol iteration, hiprandStateMRG32k3a_t* states, const complex* randTable, const int length) {
	const int index = blockDim.x * blockIdx.x + threadIdx.x;
	int sample_point, power = 1, lambda = 1;
	complex c, z, z_start, tortoise;

	for (int i = 0; i < iteration.samples_per_thread; i++) {
		// Generate sample
		c = curand_withtable(states, randTable, length);

		// Initialize complex number z and flag sample_point
		z_start.real = 0; z_start.imag = 0;

		z = z_start;
		tortoise = z;
		sample_point = 0;

		if (checkinMainBulb(c) || checkinSecondDisc(c))
			continue;

		// Judge whether a point z is escape.
		for (int j = 0; j < iteration.max_iteration; j++) {
			z = f(z, c);

			if (z.real * z.real + z.imag * z.imag > 10.0) {
				if (j >= iteration.min_iteration) {
					sample_point = 1;
				}
				break;
			}
			else if (tortoise.real == z.real && tortoise.imag == z.imag) {
				break;
			}
			else if (power == lambda + 1) {
				tortoise = z;
				power *= 2;
				lambda = 1;
			}
			lambda++;
		}

		// sampling
		if (sample_point) {
			// Initialize complex number z
			z = z_start;

			for (int j = 0; j < iteration.max_iteration; j++) {
				z = f(z, c);

				if (z.real * z.real + z.imag * z.imag > 10.0) {
					break;
				}
				else{
					draw_point(buddha, z, graph);
				}
			}
		}
	}
}



int checkImportance(const int* importance, const int i, const int j) {
	for (int dx = -1; dx < 2; dx++) {
		for (int dy = -1; dy < 2; dy++) {
			if (-1 < dx + i && dx + i < RTGRIDNUM && -1 < dy + j && dy + j < RTGRIDNUM && importance[(i + dx) + RTGRIDNUM * (j + dy)]) {
				return 1;
			}
		}
	}
	return 0;
}

unsigned long long int est_min(unsigned long long int* data, unsigned int n) {
	int length = g.w * g.h;
	unsigned long long int toReturn[10] = { data[0] };

	for (int i = 1; i < length; i++) {
		for (int j = 0; j < 10; j++) {
			if (data[i] < toReturn[j]) {
				toReturn[j] = data[i];
				break;
			}
		}
	}
	return toReturn[n];
}

unsigned long long int est_max(unsigned long long int* data, unsigned int n) {
	int length = g.w * g.h;
	unsigned long long int toReturn = data[0];

	for (int i = 1; i < length; i++) {
		if (data[i] > toReturn) {
			toReturn = data[i];
		}
	}
	return toReturn;
}

void saveImage(unsigned long long int* data, graphic g) {
	int tmp, min, max;
	FILE* fp = fopen("../../output.pgm", "wb");

	// Write header.
	fprintf(fp, "P5\n%d %d\n%d\n", g.w, g.h, 0xff);

	min = est_min(data, 1);
	max = est_max(data, 1);

	// Write pixel.
	for (int i = 0; i < g.h; i++) {
		for (int j = 0; j < g.w; j++) {
			tmp = 0xff * sqrt((data[i * g.w + j] - min) / ((double)max));
			putc(tmp, fp);
		}
	}
	 
	fclose(fp);
}


hipError_t renderImage(unsigned long long int* buddha, const graphic graph, const iterationContorol iteration) {
	const int blocks = 256 * 256, threads = 16;
	unsigned long long int* dev_buddha;
	complex* dev_randTable;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	// Initiarize random generator.
	hiprandStateMRG32k3a_t* dev_states;

	cudaStatus = hipMalloc((void**)& dev_states, blocks * threads * sizeof(hiprandStateMRG32k3a_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	initRNG <<<blocks, threads >>> (1222, dev_states);
	subend_t = clock();
	printf("Initirizing has done. (%.2f)\n", (double)(subend_t - start_t)/CLOCKS_PER_SEC);

	//Make random table.
	dim3 rtblocks = { 256, 256, 1 }, rtthreads = { RTGRIDNUM / rtblocks.x, RTGRIDNUM / rtblocks.y, 1 };
	int* dev_importance;

	cudaStatus = hipMalloc((void**)& dev_importance, RTGRIDNUM * RTGRIDNUM * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	estImportance <<<rtblocks, rtthreads >>> (dev_importance, graph, iteration);
	subend_t = clock();
	printf("Esting importance has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "estImportance launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	int* importance = (int*)malloc(RTGRIDNUM * RTGRIDNUM * sizeof(int));
	for (int i = 0; i < RTGRIDNUM * RTGRIDNUM; i++) {
		importance[i] = 0;
	}

	cudaStatus = hipMemcpy(importance, dev_importance, sizeof(int) * RTGRIDNUM * RTGRIDNUM, hipMemcpyDeviceToHost);
	hipFree(dev_importance);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}


	int sum = 0, rtindex = 0;
	complex c;

	for (int i = 0; i < RTGRIDNUM; i++) {
		for (int j = 0; j < RTGRIDNUM; j++) {
			if (checkImportance(importance, i, j))
				sum++;
		}
	}

	complex* randTable = (complex*)malloc(sizeof(complex) * sum);

	for (int i = 0; i < RTGRIDNUM; i++) {
		for (int j = 0; j < RTGRIDNUM; j++) {
			if (checkImportance(importance, i, j)) {
				c.real = -3.2 + 6.4 * i / RTGRIDNUM;
				c.imag = -3.2 + 6.4 * j / RTGRIDNUM;
				randTable[rtindex] = c;
				rtindex++;
			}
		}
	}
	subend_t = clock();
	printf("Makin random table has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

	free(importance);

	// Allocate GPU buffers for a vectors (one output).
	cudaStatus = hipMalloc((void**)& dev_buddha, g.w * g.h * sizeof(unsigned long long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& dev_randTable, sum * sizeof(complex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_buddha, buddha, g.w * g.h * sizeof(unsigned long long int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_randTable, randTable, sum * sizeof(complex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	// Compute buddhabrot.
	computeBuddhabrot <<<blocks, threads>>> (dev_buddha, graph, iteration, dev_states, dev_randTable, sum);
	subend_t = clock();
	printf("Computing buddhabrot has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeBuddhabrot launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching renderImage!\n", cudaStatus);
		goto Error;
	}
	subend_t = clock();
	printf("hipDeviceSynchronize has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);
	
	//Copy output vectors from GPU buffers to host memory.
	cudaStatus = hipMemcpy(buddha, dev_buddha, g.w * g.h * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}
	subend_t = clock();
	printf("cudaMencpy from dev_buddha to buddha has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

Error:
	hipFree(dev_states);
	hipFree(dev_buddha);
	hipFree(dev_randTable);

	free(randTable);

	return cudaStatus;
}

void set_param(int argc, char** argv) {
	// Subsitute to parameters.
	if (argc > 1) {
		for (int i = 1; i < argc; i+=2) {
			if (argv[i][1] == *"w") {
				g.w = strtol(argv[i+1], NULL, 10);
			}
			else if (argv[i][1] == *"h") {
				g.h = strtol(argv[i+1], NULL, 10);
			}
			else if (argv[i][1] == *"cr") {
				g.center.real = strtod(argv[i + 1], NULL);
			}
			else if (argv[i][1] == *"ci") {
				g.center.imag = strtod(argv[i + 1], NULL);
			}
			else if (argv[i][1] == *"s") {
				g.size = strtod(argv[i + 1], NULL);
			}
			else if (argv[i][1] == *"max") {
				iteration.max_iteration = strtol(argv[i + 1], NULL, 10);
			}
			else if (argv[i][1] == *"min") {
				iteration.min_iteration = strtol(argv[i + 1], NULL, 10);
			}
			else if (argv[i][1] == *"per") {
				iteration.samples_per_thread = strtol(argv[i + 1], NULL, 10);
			}
			else{
				fprintf(stderr, "Invalid options !");
				exit(1);
			}
		}
	}

	// Compute
	g.ratio = ((double)g.w) / g.h;
	g.dx = g.size / g.h;
	g.dy = g.size / g.h;
	g.max_real = g.center.real + 0.5 * g.size * g.ratio;
	g.max_imag = g.center.imag + 0.5 * g.size;
	g.min_real = g.center.real - 0.5 * g.size * g.ratio;
	g.min_imag = g.center.imag - 0.5 * g.size;
}


int main(int argc, char** argv)
{
	// Default value.
	g.w = WIDTH;
	g.h = HEIGHT;
	g.center.real = -0.5; // -0.15943359375;
	g.center.imag = 0.0; // 1.034150390625;
	g.size = 2.6;// 0.03125;

	iteration.samples_per_thread = 32;
	iteration.min_iteration = 0;
	iteration.max_iteration = 1000;

	set_param(argc, argv);

	printf("%f", g.ratio);

	unsigned long long int* buddha = (unsigned long long int*)malloc(sizeof(unsigned long long int) * g.w * g.h);
	if (buddha == NULL) {
		printf("Memory cannot be allocated.\n");
		free(buddha);
		return 1;
	}
	for (int i = 0; i < g.w * g.h; i++) {
		buddha[i] = 0;
	}

	start_t = clock();

	// compute and render buddhabrot.
	hipError_t cudaStatus = renderImage(buddha, g, iteration);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "renderImage failed!\n");
		return 1;
	}

	// save image of buddhabrot.
	saveImage(buddha, g);
	subend_t = clock();
	printf("Saving image has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	free(buddha);

    return 0;
}
