
#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define WIDTH 1280
#define HEIGHT 720


typedef struct {
	int w;
	int h;
	double ratio;

	double dx;
	double dy;

	double max_real;
	double min_real;
	double max_imag;
	double min_imag;
} graphic;

typedef struct {
	float real;
	float imag;
} complex;

typedef struct {
	int samples_per_thread;
	int min_iteration;
	int max_iteration;
} iterationContorol;

hipError_t renderImage(int* buddha, graphic graph, iterationContorol iteration);



__global__ void initRNG(const unsigned int seed, hiprandStateMRG32k3a_t* states) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	hiprand_init(seed, index, 0, states + index);
}

__device__ void draw_point(int* buddha, complex z, const graphic g) {
	int xnum, ynum;
	if (g.min_real < z.real && z.real < g.max_real &&
		g.min_imag < z.imag && z.imag < g.max_imag) {
		xnum = (z.real - g.min_real) / g.dx;
		ynum = (z.imag - g.min_imag) / g.dy;

		buddha[xnum + ynum * g.w] += 1;
	}
}

__device__ int checkinMainBulb(complex z) {
	float q = (z.real - 1.0 / 4.0) * (z.real - 1.0 / 4.0) + z.imag * z.imag;
	if (q * (q + (z.real - 1.0 / 4.0)) < (z.imag * z.imag) / 4.0) {
		return 1;
	}
	else {
		return 0;
	}
}

__device__ int checkinSecondDisc(complex z) {
	if ((z.real + 1) * (z.real + 1) + z.imag * z.imag < 0.25*0.25) {
		return 1;
	}
	else {
		return 0;
	}
}

__global__ void computeBuddhabrot(int* buddha, const graphic graph, iterationContorol iteration, hiprandStateMRG32k3a_t* states) {
	const int index = blockDim.x * blockIdx.x + threadIdx.x;
	int sample_point;
	complex c, z, z_tmp, z_start;

	for (int i = 0; i < iteration.samples_per_thread; i++) {
		// Generate sample
		c.real = -3 + 6*hiprand_uniform(&states[index]);
		c.imag = -3 + 6*hiprand_uniform(&states[index]);


		// Initialize complex number z and flag sample_point
		z_start.real = 0; z_start.imag = 0;

		z = z_start;
		sample_point = 0;

		if (checkinMainBulb(c) || checkinSecondDisc(c))
			continue;

		// Judge whether a point z is escape.
		for (int j = 0; j < iteration.max_iteration; j++) {
			z_tmp.real = z.real * z.real - z.imag * z.imag + c.real;
			z_tmp.imag = 2 * z.real * z.imag + c.imag;
			z = z_tmp;

			if (z.real * z.real + z.imag * z.imag > 16.0) {
				if (j > iteration.min_iteration) {
					sample_point = 1;
				}
				break;
			}
		}

		// sampling
		if (sample_point) {
			// Initialize complex number z
			z = z_start;

			for (int j = 0; j < iteration.max_iteration; j++) {
				z_tmp.real = (z.real * z.real - z.imag * z.imag) + c.real;
				z_tmp.imag = 2 * z.real * z.imag + c.imag;
				z = z_tmp;

				if (z.real * z.real + z.imag * z.imag > 16.0) {
					break;
				}
				else{
					draw_point(buddha, z, graph);
				}
			}
		}
	}
}



int est_min(int* data, unsigned int n) {
	int length = WIDTH * HEIGHT;
	int toReturn[10] = { data[0] };

	for (int i = 1; i < length; i++) {
		for (int j = 0; j < 10; j++) {
			if (data[i] < toReturn[j]) {
				toReturn[j] = data[i];
				break;
			}
		}
	}
	return toReturn[n];
}

int est_max(int* data, unsigned int n) {
	int length = WIDTH * HEIGHT;
	int toReturn = data[0];

	for (int i = 1; i < length; i++) {
		if (data[i] > toReturn) {
			toReturn = data[i];
		}
	}
	return toReturn;
}

void saveImage(int* data, graphic g) {
	int tmp, min, max;
	FILE* fp = fopen("../../output.pgm", "wb");

	// Write header.
	fprintf(fp, "P2\n%d %d\n%d\n", g.w, g.h, 0xffff);

	min = est_min(data, 1);
	max = est_max(data, 1);

	// Write pixel.
	for (int i = 0; i < g.h; i++) {
		for (int j = 0; j < g.w; j++) {
			tmp = 0xffff * (data[i * g.w + j] - min) / ((double)max);
			fprintf(fp, "%d ", tmp);
		}
		fprintf(fp, "\n");
	}
	 
	fclose(fp);
}



hipError_t renderImage(int* buddha, graphic graph, iterationContorol iteration) {
	hiprandStateMRG32k3a_t* dev_states;
	int* dev_buddha;

	hipError_t cudaStatus;
	

	const int blocks = 256*256;
	const int threads = 16;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for a vectors (one output).
	cudaStatus = hipMalloc((void**)& dev_buddha, WIDTH * HEIGHT * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& dev_states, blocks * threads * sizeof(hiprandStateMRG32k3a_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_buddha, buddha, WIDTH * HEIGHT * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Initialize random generator.
	initRNG <<<blocks, threads>>> (1222, dev_states);
	
	// Compute buddhabrot.
	computeBuddhabrot <<<blocks, threads>>> (dev_buddha, graph, iteration, dev_states);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	
	//Copy output vectors from GPU buffers to host memory.
	cudaStatus = hipMemcpy(buddha, dev_buddha, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_buddha);
	hipFree(dev_states);
	
	return cudaStatus;
}



int main()
{
	complex center;
	center.real = -0.5;
	center.imag = 0.0;

	float size = 2.0;

	graphic g;
	g.w = WIDTH;
	g.h = HEIGHT;
	g.ratio = ((double)WIDTH) / HEIGHT;
	g.dx = size / g.h;
	g.dy = size / g.h;
	g.max_real = center.real + 0.5 * size * g.ratio;
	g.max_imag = center.imag + 0.5 * size;
	g.min_real = center.real - 0.5 * size * g.ratio;
	g.min_imag = center.imag - 0.5 * size;

	iterationContorol iteration;
	iteration.samples_per_thread = 100;
	iteration.min_iteration = 1;
	iteration.max_iteration = 400;

	int* buddha = (int*)malloc(sizeof(int) * WIDTH * HEIGHT);
	if (buddha == NULL) {
		printf("Memory cannot be allocated.");
		free(buddha);
		return 1;
	}
	for (int i = 0; i < WIDTH * HEIGHT; i++) {
		buddha[i] = 0;
	}

	// compute and render buddhabrot.
	hipError_t cudaStatus = renderImage(buddha, g, iteration);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "renderImage failed!");
		return 1;
	}

	// save image of buddhabrot.
	buddha[1] = 1;
	saveImage(buddha, g);

	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	free(buddha);

    return 0;
}
