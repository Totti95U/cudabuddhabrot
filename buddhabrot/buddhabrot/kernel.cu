#include "hip/hip_runtime.h"
/*
To delete "warning C4819"
1. Open property of buddhabrot project.
2. Open [CUDA C/C++]/[Command Line].
3. Write "-Xcompiler -wd4819" in additional options.
*/


#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define WIDTH 1280
#define HEIGHT 720
#define RTGRIDNUM 2048

typedef struct {
	int w;
	int h;
	double ratio;

	double dx;
	double dy;

	double max_real;
	double min_real;
	double max_imag;
	double min_imag;
} graphic;

typedef struct {
	float real;
	float imag;
} complex;

typedef struct {
	int samples_per_thread;
	int min_iteration;
	int max_iteration;
} iterationContorol;


hipError_t renderImage(unsigned long long int* buddha, const graphic graph, const iterationContorol iteration);

__device__ complex f(complex z, complex c) {
	complex toReturn;
	toReturn.real = z.real * z.real - z.imag * z.imag + c.real;
	toReturn.imag = 2 * z.real * z.imag + c.imag;
	return toReturn;
}

__global__ void initRNG(const unsigned int seed, hiprandStateMRG32k3a_t* states) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	hiprand_init(seed, index, 0, states + index);
}

__device__ int checkinWindow(complex z, graphic graph) {
	if (graph.min_real < z.real && z.real < graph.max_real &&
		graph.min_imag < z.imag && z.imag < graph.max_imag) {
		return 1;
	}
	return 0;
}

__device__ int checkinMainBulb(complex z) {
	float q = (z.real - 1.0 / 4.0) * (z.real - 1.0 / 4.0) + z.imag * z.imag;
	if (q * (q + (z.real - 1.0 / 4.0)) < (z.imag * z.imag) / 4.0) {
		return 1;
	}
	else {
		return 0;
	}
}

__device__ int checkinSecondDisc(complex z) {
	if ((z.real + 1) * (z.real + 1) + z.imag * z.imag < 0.25 * 0.25) {
		return 1;
	}
	else {
		return 0;
	}
}

__global__ void estImportance(int* importance, graphic graph, iterationContorol iteration) {
	int indexx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int indexy = (blockIdx.y * blockDim.y) + threadIdx.y;
	complex c, z;

	// Initiarize complex num c , z and int importance.
	c.real = -3.2 + 6.4 * indexx / RTGRIDNUM;
	c.imag = -3.2 + 6.4 * indexy / RTGRIDNUM;
	z.real = 0.0; z.imag = 0.0;
	importance[indexx + indexy * RTGRIDNUM] = 0;
	/*
	if (checkinMainBulb(c) || checkinSecondDisc(c)) {
	 	importance[indexx + indexy * RTGRIDNUM] = 0;
		return;
	}
	*/

	for (int i = 0; i < iteration.max_iteration; i++) {
		z = f(z, c);
		if (z.real * z.real + z.imag * z.imag > 10.0) {
			return;
		}
		else if (i == iteration.max_iteration - 1) {
			importance[indexx + indexy * RTGRIDNUM] = 0;
			return;
		}
		else if (checkinWindow(z, graph) && i >= iteration.min_iteration) {
			importance[indexx + indexy * RTGRIDNUM] = 1;
		}
	}
}

__device__ void draw_point(unsigned long long int* buddha, complex z, const graphic g) {
	int xnum, ynum;
	if (checkinWindow(z, g)) {
		xnum = (z.real - g.min_real) / g.dx;
		ynum = g.h - (z.imag - g.min_imag) / g.dy;

		buddha[xnum + ynum * g.w] += 1;
	}
}

__device__ complex curand_withtable(hiprandStateMRG32k3a_t* state, const complex* randTable, const int length) {
	complex toReturn;
	const int index = blockDim.x * blockIdx.x + threadIdx.x;

	int t_index = hiprand(&state[index]) % length;
	toReturn = randTable[t_index];
	toReturn.real += (-3.2 + 6.4 * hiprand_uniform(&state[index])) / RTGRIDNUM;
	toReturn.imag += (-3.2 + 6.4 * hiprand_uniform(&state[index])) / RTGRIDNUM;
	return toReturn;
}

__global__ void computeBuddhabrot(unsigned long long int* buddha, const graphic graph, const iterationContorol iteration, hiprandStateMRG32k3a_t* states, const complex* randTable, const int length) {
	const int index = blockDim.x * blockIdx.x + threadIdx.x;
	int sample_point, power = 1, lambda = 1;
	complex c, z, z_start, tortoise;

	for (int i = 0; i < iteration.samples_per_thread; i++) {
		// Generate sample
		c = curand_withtable(states, randTable, length);

		// Initialize complex number z and flag sample_point
		z_start.real = 0; z_start.imag = 0;

		z = z_start;
		tortoise = z;
		sample_point = 0;

		/*
		if (checkinMainBulb(c) || checkinSecondDisc(c))
			continue;
		*/

		// Judge whether a point z is escape.
		for (int j = 0; j < iteration.max_iteration; j++) {
			z = f(z, c);

			if (z.real * z.real + z.imag * z.imag > 10.0) {
				if (j >= iteration.min_iteration) {
					sample_point = 1;
				}
				break;
			}
			else if (tortoise.real == z.real && tortoise.imag == z.imag) {
				break;
			}
			else if (power == lambda + 1) {
				tortoise = z;
				power *= 2;
				lambda = 1;
			}
			lambda++;
		}

		// sampling
		if (sample_point) {
			// Initialize complex number z
			z = z_start;

			for (int j = 0; j < iteration.max_iteration; j++) {
				z = f(z, c);

				if (z.real * z.real + z.imag * z.imag > 10.0) {
					break;
				}
				else{
					draw_point(buddha, z, graph);
				}
			}
		}
	}
}



int checkImportance(const int* importance, const int i, const int j) {
	for (int dx = -1; dx < 2; dx++) {
		for (int dy = -1; dy < 2; dy++) {
			if (-1 < dx + i && dx + i < RTGRIDNUM && -1 < dy + j && dy + j < RTGRIDNUM && importance[(i + dx) + RTGRIDNUM * (j + dy)]) {
				return 1;
			}
		}
	}
	return 0;
}

unsigned long long int est_min(unsigned long long int* data, unsigned int n) {
	int length = WIDTH * HEIGHT;
	unsigned long long int toReturn[10] = { data[0] };

	for (int i = 1; i < length; i++) {
		for (int j = 0; j < 10; j++) {
			if (data[i] < toReturn[j]) {
				toReturn[j] = data[i];
				break;
			}
		}
	}
	return toReturn[n];
}

unsigned long long int est_max(unsigned long long int* data, unsigned int n) {
	int length = WIDTH * HEIGHT;
	unsigned long long int toReturn = data[0];

	for (int i = 1; i < length; i++) {
		if (data[i] > toReturn) {
			toReturn = data[i];
		}
	}
	return toReturn;
}

void saveImage(unsigned long long int* data, graphic g) {
	int tmp, min, max;
	FILE* fp = fopen("../../output.pgm", "wb");

	// Write header.
	fprintf(fp, "P5\n%d %d\n%d\n", g.w, g.h, 0xff);

	min = est_min(data, 1);
	max = est_max(data, 1);

	// Write pixel.
	for (int i = 0; i < g.h; i++) {
		for (int j = 0; j < g.w; j++) {
			tmp = 0xff * sqrt((data[i * g.w + j] - min) / ((double)max));
			putc(tmp, fp);
		}
	}
	 
	fclose(fp);
}


hipError_t renderImage(unsigned long long int* buddha, const graphic graph, const iterationContorol iteration) {
	const int blocks = 256 * 256, threads = 16;
	unsigned long long int* dev_buddha;
	complex* dev_randTable;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	// Initiarize random generator.
	hiprandStateMRG32k3a_t* dev_states;

	cudaStatus = hipMalloc((void**)& dev_states, blocks * threads * sizeof(hiprandStateMRG32k3a_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	initRNG << <blocks, threads >> > (1222, dev_states);

	//Make random table.
	dim3 rtblocks = { 256, 256, 1 }, rtthreads = { RTGRIDNUM / rtblocks.x, RTGRIDNUM / rtblocks.y, 1 };
	int* dev_importance;

	cudaStatus = hipMalloc((void**)& dev_importance, RTGRIDNUM * RTGRIDNUM * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	estImportance <<<rtblocks, rtthreads >>> (dev_importance, graph, iteration);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "estImportance launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	int* importance = (int*)malloc(RTGRIDNUM * RTGRIDNUM * sizeof(int));
	for (int i = 0; i < RTGRIDNUM * RTGRIDNUM; i++) {
		importance[i] = 0;
	}

	cudaStatus = hipMemcpy(importance, dev_importance, sizeof(int) * RTGRIDNUM * RTGRIDNUM, hipMemcpyDeviceToHost);
	hipFree(dev_importance);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	int sum = 0, rtindex = 0;
	complex c;

	for (int i = 0; i < RTGRIDNUM; i++) {
		for (int j = 0; j < RTGRIDNUM; j++) {
			if (checkImportance(importance, i, j))
				sum++;
		}
	}

	complex* randTable = (complex*)malloc(sizeof(complex) * sum);
	printf("randTable malloced. (length: %d)\n", sum);

	for (int i = 0; i < RTGRIDNUM; i++) {
		for (int j = 0; j < RTGRIDNUM; j++) {
			if (checkImportance(importance, i, j)) {
				c.real = -3.2 + 6.4 * i / RTGRIDNUM;
				c.imag = -3.2 + 6.4 * j / RTGRIDNUM;
				randTable[rtindex] = c;
				rtindex++;
			}
		}
	}

	free(importance);

	// Allocate GPU buffers for a vectors (one output).
	cudaStatus = hipMalloc((void**)& dev_buddha, WIDTH * HEIGHT * sizeof(unsigned long long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& dev_randTable, sum * sizeof(complex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_buddha, buddha, WIDTH * HEIGHT * sizeof(unsigned long long int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_randTable, randTable, sum * sizeof(complex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	// Compute buddhabrot.
	computeBuddhabrot <<<blocks, threads>>> (dev_buddha, graph, iteration, dev_states, dev_randTable, sum);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeBuddhabrot launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching renderImage!\n", cudaStatus);
		goto Error;
	}
	
	//Copy output vectors from GPU buffers to host memory.
	cudaStatus = hipMemcpy(buddha, dev_buddha, WIDTH * HEIGHT * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

Error:
	hipFree(dev_states);
	hipFree(dev_buddha);
	hipFree(dev_randTable);

	free(randTable);

	return cudaStatus;
}



int main()
{
	complex center;
	center.real = -0.5; // -0.15943359375;
	center.imag = 0.0; // 1.034150390625;

	double size = 2.6;// 0.03125;// 2.6;

	graphic g;
	g.w = WIDTH;
	g.h = HEIGHT;
	g.ratio = ((double)WIDTH) / HEIGHT;
	g.dx = size / g.h;
	g.dy = size / g.h;
	g.max_real = center.real + 0.5 * size * g.ratio;
	g.max_imag = center.imag + 0.5 * size;
	g.min_real = center.real - 0.5 * size * g.ratio;
	g.min_imag = center.imag - 0.5 * size;

	iterationContorol iteration;
	iteration.samples_per_thread = 256;
	iteration.min_iteration = 0;
	iteration.max_iteration = 20;


	unsigned long long int* buddha = (unsigned long long int*)malloc(sizeof(unsigned long long int) * WIDTH * HEIGHT);
	if (buddha == NULL) {
		printf("Memory cannot be allocated.\n");
		free(buddha);
		return 1;
	}
	for (int i = 0; i < WIDTH * HEIGHT; i++) {
		buddha[i] = 0;
	}

	// compute and render buddhabrot.
	hipError_t cudaStatus = renderImage(buddha, g, iteration);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "renderImage failed!\n");
		return 1;
	}

	// save image of buddhabrot.
	saveImage(buddha, g);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	free(buddha);

    return 0;
}
