#include "hip/hip_runtime.h"
/*
To delete "warning C4819"
1. Open property of buddhabrot project.
2. Open [CUDA C/C++]/[Command Line].
3. Write "-Xcompiler -wd4819" in additional options.
*/


#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define WIDTH 1280
#define HEIGHT 720


typedef struct {
	int w;
	int h;
	double ratio;

	double dx;
	double dy;

	double max_real;
	double min_real;
	double max_imag;
	double min_imag;
} graphic;

typedef struct {
	float real;
	float imag;
} complex;

typedef struct {
	int samples_per_thread;
	int min_iteration;
	int max_iteration;
} iterationContorol;


hipError_t renderImage(unsigned long long int* buddha, const graphic graph, const iterationContorol iteration);

__global__ void initRNG(const unsigned int seed, hiprandStateMRG32k3a_t* states) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	hiprand_init(seed, index, 0, states + index);
}

__device__ int checkinWindow(complex z, graphic graph) {
	if (graph.min_real < z.real && z.real < graph.max_real &&
		graph.min_imag < z.imag && z.imag < graph.max_imag) {
		return 1;
	}
	return 0;
}

__device__ int checkinMainBulb(complex z) {
	float q = (z.real - 1.0 / 4.0) * (z.real - 1.0 / 4.0) + z.imag * z.imag;
	if (q * (q + (z.real - 1.0 / 4.0)) < (z.imag * z.imag) / 4.0) {
		return 1;
	}
	else {
		return 0;
	}
}

__device__ int checkinSecondDisc(complex z) {
	if ((z.real + 1) * (z.real + 1) + z.imag * z.imag < 0.25 * 0.25) {
		return 1;
	}
	else {
		return 0;
	}
}

__global__ void estImportance(int* importance, graphic graph, iterationContorol iteration, const int gridnum) {
	int indexx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int indexy = (blockIdx.y * blockDim.y) + threadIdx.y;
	complex c, z_tmp, z;

	// Initiarize complex num c , z and int importance.
	c.real = -3.2 + 6.4 * indexx / gridnum;
	c.imag = -3.2 + 6.4 * indexy / gridnum;
	z.real = 0.0; z.imag = 0.0;
	importance[indexx + indexy * gridnum] = 0;

	if (checkinMainBulb(c) || checkinSecondDisc(c)) {
		importance[indexx + indexy * gridnum] = 0;
		return;
	}

	for (int i = 0; i < iteration.max_iteration; i++) {
		z_tmp.real = z.real * z.real - z.imag * z.imag + c.real;
		z_tmp.imag = 2 * z.real * z.imag + c.imag;
		z = z_tmp;
		if (z.real * z.real + z.imag * z.imag > 10.0) {
			return;
		}
		else if (i == iteration.max_iteration - 1) {
			importance[indexx + indexy * gridnum] = 0;
			return;
		}
		else if (checkinWindow(z, graph) && i >= iteration.min_iteration) {
			importance[indexx + indexy * gridnum] = 1;
		}
	}
}

__device__ void draw_point(unsigned long long int* buddha, complex z, const graphic g) {
	int xnum, ynum;
	if (checkinWindow(z, g)) {
		xnum = (z.real - g.min_real) / g.dx;
		ynum = (z.imag - g.min_imag) / g.dy;

		buddha[xnum + ynum * g.w] += 1;
	}
}

__device__ complex curand_withtable(hiprandStateMRG32k3a_t* state, const complex* randTable, const int length, const int num) {
	complex toReturn;
	const int index = blockDim.x * blockIdx.x + threadIdx.x;

	int t_index = hiprand(&state[index]) % length;
	toReturn = randTable[t_index];
	toReturn.real += (-3.2 + 6.4 * hiprand_uniform(&state[index])) / 2048;
	toReturn.imag += (-3.2 + 6.4 * hiprand_uniform(&state[index])) / 2048;
	return toReturn;
}

__global__ void computeBuddhabrot(unsigned long long int* buddha, const graphic graph, const iterationContorol iteration, hiprandStateMRG32k3a_t* states, const complex* randTable, const int length) {
	const int index = blockDim.x * blockIdx.x + threadIdx.x;
	int sample_point, power = 1, lambda = 1;
	complex c, z, z_tmp, z_start, tortoise;

	for (int i = 0; i < iteration.samples_per_thread; i++) {
		// Generate sample
		c = curand_withtable(states, randTable, length, i);

		// Initialize complex number z and flag sample_point
		z_start.real = 0; z_start.imag = 0;

		z = z_start;
		tortoise = z;
		sample_point = 0;

		if (checkinMainBulb(c) || checkinSecondDisc(c))
			continue;

		// Judge whether a point z is escape.
		for (int j = 0; j < iteration.max_iteration; j++) {
			z_tmp.real = z.real * z.real - z.imag * z.imag + c.real;
			z_tmp.imag = 2 * z.real * z.imag + c.imag;
			z = z_tmp;

			if (z.real * z.real + z.imag * z.imag > 10.0) {
				if (j >= iteration.min_iteration) {
					sample_point = 1;
				}
				break;
			}
			else if (tortoise.real == z.real && tortoise.imag == z.imag) {
				break;
			}
			else if (power == lambda + 1) {
				tortoise = z;
				power *= 2;
				lambda = 1;
			}
			lambda++;
		}

		// sampling
		if (sample_point) {
			// Initialize complex number z
			z = z_start;

			for (int j = 0; j < iteration.max_iteration; j++) {
				z_tmp.real = (z.real * z.real - z.imag * z.imag) + c.real;
				z_tmp.imag = 2 * z.real * z.imag + c.imag;
				z = z_tmp;

				if (z.real * z.real + z.imag * z.imag > 10.0) {
					break;
				}
				else{
					draw_point(buddha, z, graph);
				}
			}
		}
	}
}



int checkImportance(const int* importance, const int i, const int j, const int gridnum) {
	if (importance[i + j*gridnum])
		return 1;

	else if (i > 0) {
		if (importance[i - 1 + j * gridnum])
			return 1;
	}
	else if (i < gridnum) {
		if (importance[i + 1 + j * gridnum])
			return 1;
	}
	else if (j > 0) {
		if (importance[i + (j - 1) * gridnum])
			return 1;
	}
	else if (j < gridnum) {
		if (importance[i + (j + 1) * gridnum])
			return 1;
	}
	return 0;
}

unsigned long long int est_min(unsigned long long int* data, unsigned int n) {
	int length = WIDTH * HEIGHT;
	unsigned long long int toReturn[10] = { data[0] };

	for (int i = 1; i < length; i++) {
		for (int j = 0; j < 10; j++) {
			if (data[i] < toReturn[j]) {
				toReturn[j] = data[i];
				break;
			}
		}
	}
	return toReturn[n];
}

unsigned long long int est_max(unsigned long long int* data, unsigned int n) {
	int length = WIDTH * HEIGHT;
	unsigned long long int toReturn = data[0];

	for (int i = 1; i < length; i++) {
		if (data[i] > toReturn) {
			toReturn = data[i];
		}
	}
	return toReturn;
}

void saveImage(unsigned long long int* data, graphic g) {
	int tmp, min, max;
	FILE* fp = fopen("../../output.pgm", "wb");

	// Write header.
	fprintf(fp, "P2\n%d %d\n%d\n", g.w, g.h, 0xffff);

	min = est_min(data, 1);
	max = est_max(data, 1);

	// Write pixel.
	for (int i = 0; i < g.h; i++) {
		for (int j = 0; j < g.w; j++) {
			tmp = 0xffff * sqrt((data[i * g.w + j] - min) / ((double)max));
			fprintf(fp, "%d ", tmp);
		}
		fprintf(fp, "\n");
	}
	 
	fclose(fp);
}


hipError_t renderImage(unsigned long long int* buddha, const graphic graph, const iterationContorol iteration) {
	const int blocks = 256 * 256, threads = 16;
	unsigned long long int* dev_buddha;
	complex* dev_randTable;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	// Initiarize random generator.
	hiprandStateMRG32k3a_t* dev_states;

	cudaStatus = hipMalloc((void**)& dev_states, blocks * threads * sizeof(hiprandStateMRG32k3a_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	initRNG << <blocks, threads >> > (1222, dev_states);

	//Make random table.
	int rtGridnum = 2048;
	dim3 rtblocks = { 256, 256, 1 }, rtthreads = { rtGridnum/rtblocks.x, rtGridnum / rtblocks.y, 1 };
	int* dev_importance;

	cudaStatus = hipMalloc((void**)& dev_importance, rtGridnum * rtGridnum * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	estImportance <<<rtblocks, rtthreads >>> (dev_importance, graph, iteration, rtGridnum);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "estImportance launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	int* importance = (int*)malloc(rtGridnum * rtGridnum * sizeof(int));
	for (int i = 0; i < rtGridnum * rtGridnum; i++) {
		importance[i] = 0;
	}

	cudaStatus = hipMemcpy(importance, dev_importance, sizeof(int) * rtGridnum * rtGridnum, hipMemcpyDeviceToHost);
	hipFree(dev_importance);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	int sum = 0, rtindex = 0;
	complex c;

	for (int i = 0; i < rtGridnum; i++) {
		for (int j = 0; j < rtGridnum; j++) {
			if (checkImportance(importance, i, j, rtGridnum))
				sum++;
		}
	}

	complex* randTable = (complex*)malloc(sizeof(complex) * sum);
	printf("randTable malloced. (length: %d)\n", sum);

	for (int i = 0; i < rtGridnum; i++) {
		for (int j = 0; j < rtGridnum; j++) {
			if (checkImportance(importance, i, j, rtGridnum)) {
				c.real = -3.2 + 6.4 * i / rtGridnum;
				c.imag = -3.2 + 6.4 * j / rtGridnum;
				randTable[rtindex] = c;
				rtindex++;
			}
		}
	}

	free(importance);

	// Allocate GPU buffers for a vectors (one output).
	cudaStatus = hipMalloc((void**)& dev_buddha, WIDTH * HEIGHT * sizeof(unsigned long long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& dev_randTable, sum * sizeof(complex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_buddha, buddha, WIDTH * HEIGHT * sizeof(unsigned long long int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_randTable, randTable, sum * sizeof(complex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	// Compute buddhabrot.
	computeBuddhabrot <<<blocks, threads>>> (dev_buddha, graph, iteration, dev_states, dev_randTable, sum);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeBuddhabrot launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching renderImage!\n", cudaStatus);
		goto Error;
	}
	
	//Copy output vectors from GPU buffers to host memory.
	cudaStatus = hipMemcpy(buddha, dev_buddha, WIDTH * HEIGHT * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

Error:
	hipFree(dev_states);
	hipFree(dev_buddha);
	hipFree(dev_randTable);

	free(randTable);

	return cudaStatus;
}



int main()
{
	complex center;
	center.real = -0.15943359375; // -0.5;
	center.imag = 1.034150390625; // 0.0;

	double size = 0.03125; // 2.6;

	graphic g;
	g.w = WIDTH;
	g.h = HEIGHT;
	g.ratio = ((double)WIDTH) / HEIGHT;
	g.dx = size / g.h;
	g.dy = size / g.h;
	g.max_real = center.real + 0.5 * size * g.ratio;
	g.max_imag = center.imag + 0.5 * size;
	g.min_real = center.real - 0.5 * size * g.ratio;
	g.min_imag = center.imag - 0.5 * size;

	iterationContorol iteration;
	iteration.samples_per_thread = 128;
	iteration.min_iteration = 0;
	iteration.max_iteration = 1000;


	unsigned long long int* buddha = (unsigned long long int*)malloc(sizeof(unsigned long long int) * WIDTH * HEIGHT);
	if (buddha == NULL) {
		printf("Memory cannot be allocated.\n");
		free(buddha);
		return 1;
	}
	for (int i = 0; i < WIDTH * HEIGHT; i++) {
		buddha[i] = 0;
	}

	// compute and render buddhabrot.
	hipError_t cudaStatus = renderImage(buddha, g, iteration);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "renderImage failed!\n");
		return 1;
	}

	// save image of buddhabrot.
	saveImage(buddha, g);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	free(buddha);

    return 0;
}
