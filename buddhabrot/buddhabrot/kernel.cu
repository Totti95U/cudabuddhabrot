#include "hip/hip_runtime.h"
/*
To delete "warning C4819"
1. Open property of buddhabrot project.
2. Open [CUDA C/C++]/[Command Line].
3. Write "-Xcompiler -wd4819" in additional options.
*/


#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "hip/device_functions.h"
#include <math_functions.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#ifdef __BREAK_ME__
#include <math_functions.h>
#else
#include <hip/hip_runtime.h>
#endif

#define WIDTH 1280
#define HEIGHT 720
#define RTGRIDNUM 1024

typedef struct {
	float real;
	float imag;
} complex;

typedef struct {
	int w;
	int h;
	double ratio;

	float dx;
	float dy;

	complex center;
	float size;
	float max_real;
	float min_real;
	float max_imag;
	float min_imag;

	double gamma;
} graphic;

typedef struct {
	int samples_per_thread;
	int min_iteration;
	int max_iteration;
} iterationContorol;

typedef struct {
	int axi1;
	int axi2;
	float angl;
	float RotMat[16];
} rotationContorol;

// global variances
clock_t start_t, subend_t;

graphic g;
iterationContorol iteration;
rotationContorol rotation[6];
int rotation_axis[6*2] = { 0, 1, 1, 2, 2, 3, 3, 0, 1, 3, 0, 2 };

float RotationMatrix[16] = { 0 };

hipError_t renderImage(unsigned long long int* buddha);

__device__ complex f(complex z, complex c) {
	complex toReturn;
	toReturn.real = z.real * z.real - z.imag * z.imag + c.real;
	toReturn.imag = 2 * z.real * z.imag + c.imag;
	return toReturn;
}

__global__ void initRNG(const unsigned int seed, hiprandStateMRG32k3a_t* states) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	// standard initirized
	// hiprand_init(seed, index, 0, &states[index]);

	//fast initirized
	hiprand_init((seed << 20) + index, 0, 0, &states[index]);
}

__device__ int checkinWindow(complex z, graphic g) {
	if (g.min_real < z.real && z.real < g.max_real &&
		g.min_imag < z.imag && z.imag < g.max_imag) {
		return 1;
	}
	return 0;
}

__device__ int checkinMainBulb(complex z) {
	float q = (z.real - 1.0f / 4.0f) * (z.real - 1.0f / 4.0f) + z.imag * z.imag;
	if (q * (q + (z.real - 1.0f / 4.0f)) < (z.imag * z.imag) / 4.0f) {
		return 1;
	}
	else {
		return 0;
	}
}

__device__ int checkinSecondDisc(complex z) {
	if ((z.real + 1) * (z.real + 1) + z.imag * z.imag < 0.25f * 0.25f) {
		return 1;
	}
	else {
		return 0;
	}
}

__device__ void rot4d(const float* RotMat, complex* z, const complex* c) {
	float vect[4] = { z->real, z->imag, c->real, c->imag };

	z->real = 0.0f;
	z->imag = 0.0f;
	// c->real = 0.0f;
	// c->imag = 0.0f;

	for (int i = 0; i < 4; i++) {
		z->real += RotMat[i] * vect[i];
		z->imag += RotMat[i + 4] * vect[i];
		// c->real += RotMat[i + 8] * vect[i];
		// c->imag += RotMat[i + 12] * vect[i];
	}
}

__global__ void estImportance(int* importance, const graphic g, const iterationContorol iteration, const float* RotMat) {
	int indexx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int indexy = (blockIdx.y * blockDim.y) + threadIdx.y;
	complex c, z, rotated_z, rotated_c;

	// Initiarize complex num c , z and int importance.
	c.real = -3.2f + 6.4f * indexx / RTGRIDNUM;
	c.imag = -3.2f + 6.4f * indexy / RTGRIDNUM;
	z.real = 0.0f; z.imag = 0.0f;
	importance[indexx + indexy * RTGRIDNUM] = 0;

	if (checkinMainBulb(c) || checkinSecondDisc(c)) {
	 	importance[indexx + indexy * RTGRIDNUM] = 0;
		return;
	}

	for (int i = 0; i < iteration.max_iteration; i++) {
		z = f(z, c);
		if (z.real * z.real + z.imag * z.imag > 10.0f) {
			return;
		}
		else if (i == iteration.max_iteration - 1) {
			importance[indexx + indexy * RTGRIDNUM] = 0;
			return;
		}
		else if (i >= iteration.min_iteration) {
			rotated_z = z; //rotated_c = c;
			for (int i = 0; i < 6; i++) {
				rot4d(RotMat, &rotated_z, &c);
			}
			if (checkinWindow(rotated_z, g))
				importance[indexx + indexy * RTGRIDNUM] = 1;
		}
	}
}

__device__ void draw_point(unsigned long long int* buddha, complex z, graphic g) {
	int xnum, ynum;
	if (checkinWindow(z, g)) {
		xnum = (z.real - g.min_real) / g.dx;
		ynum = g.h - (z.imag - g.min_imag) / g.dy;

		buddha[xnum + ynum * g.w] += 1;
	}
}

__device__ complex curand_withtable(hiprandStateMRG32k3a_t* state, const complex* randTable, const int length) {
	complex toReturn;
	const int index = blockDim.x * blockIdx.x + threadIdx.x;

	int t_index = hiprand(&state[index]) % length;
	toReturn = randTable[t_index];
	toReturn.real += (-3.2f + 6.4f * hiprand_uniform(&state[index])) / RTGRIDNUM;
	toReturn.imag += (-3.2f + 6.4f * hiprand_uniform(&state[index])) / RTGRIDNUM;
	return toReturn;
}

__global__ void computeBuddhabrot(unsigned long long int* buddha, graphic g, iterationContorol iteration, float* RotMat, hiprandStateMRG32k3a_t* states, const complex* randTable, const int length) {
	const int index = blockDim.x * blockIdx.x + threadIdx.x;
	int sample_point, power = 1, lambda = 1;
	complex c, z, z_start, tortoise, rotated_z, rotated_c;

	for (int i = 0; i < iteration.samples_per_thread; i++) {
		// Generate sample
		c = curand_withtable(states, randTable, length);

		// Initialize complex number z and flag sample_point
		z_start.real = 0; z_start.imag = 0;

		z = z_start;
		tortoise = z;
		sample_point = 0;

		if (checkinMainBulb(c) || checkinSecondDisc(c))
			continue;

		// Judge whether a point z is escape.
		for (int j = 0; j < iteration.max_iteration; j++) {
			z = f(z, c);

			if (z.real * z.real + z.imag * z.imag > 10.0f) {
				if (j >= iteration.min_iteration) {
					sample_point = 1;
				}
				break;
			}
			else if (tortoise.real == z.real && tortoise.imag == z.imag) {
				break;
			}
			else if (power == lambda + 1) {
				tortoise = z;
				power *= 2;
				lambda = 1;
			}
			lambda++;
		}

		// sampling
		if (sample_point) {
			// Initialize complex number z
			z = z_start;

			for (int j = 0; j < iteration.max_iteration; j++) {
				z = f(z, c);

				if (z.real * z.real + z.imag * z.imag > 10.0f) {
					break;
				}
				else{
					rotated_z = z; //rotated_c = c;
					for (int i = 0; i < 6; i++) {
						rot4d(RotMat, &rotated_z, &c);
					}
					draw_point(buddha, rotated_z, g);
				}
			}
		}
	}
}



int checkImportance(const int* importance, const int i, const int j) {
	for (int dx = -1; dx < 2; dx++) {
		for (int dy = -1; dy < 2; dy++) {
			if (-1 < dx + i && dx + i < RTGRIDNUM && -1 < dy + j && dy + j < RTGRIDNUM && importance[(i + dx) + RTGRIDNUM * (j + dy)]) {
				return 1;
			}
		}
	}
	return 0;
}

unsigned long long int est_min(unsigned long long int* data, unsigned int n) {
	int length = g.w * g.h;
	unsigned long long int toReturn[10] = { data[0] };

	for (int i = 1; i < length; i++) {
		for (int j = 0; j < 10; j++) {
			if (data[i] < toReturn[j]) {
				toReturn[j] = data[i];
				break;
			}
		}
	}
	return toReturn[n];
}

unsigned long long int est_max(unsigned long long int* data, unsigned int n) {
	int length = g.w * g.h;
	unsigned long long int toReturn = data[0];

	for (int i = 1; i < length; i++) {
		if (data[i] > toReturn) {
			toReturn = data[i];
		}
	}
	return toReturn;
}

void saveImage(unsigned long long int* data) {
	unsigned long long int tmp, min, max;
	FILE* fp = fopen("../../output.pgm", "wb");

	// Write header.
	fprintf(fp, "P5\n%d %d\n%d\n", g.w, g.h, 0xff);

	min = est_min(data, 1);
	max = est_max(data, 1);

	// Write pixel.
	for (int i = 0; i < g.h; i++) {
		for (int j = 0; j < g.w; j++) {
			tmp = 0xff * pow((double) (data[i * g.w + j] - min) / max, 1/g.gamma);
			putc(tmp, fp);
		}
	}
	 
	fclose(fp);
}


hipError_t renderImage(unsigned long long int* buddha) {
	const int blocks = 256 * 256, threads = 512;
	unsigned long long int* dev_buddha;
	complex* dev_randTable;
	float* dev_RotationMatrix;

	dim3 rtblocks = { 256, 256, 1 }, rtthreads = { RTGRIDNUM / rtblocks.x, RTGRIDNUM / rtblocks.y, 1 };
	int* dev_importance;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	// Initiarize random generator.
	hiprandStateMRG32k3a_t* dev_states;

	cudaStatus = hipMalloc((void**)& dev_states, blocks * threads * sizeof(hiprandStateMRG32k3a_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	initRNG <<<blocks, threads >>> (1222, dev_states);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "initRNG launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	subend_t = clock();
	printf("Initirizing has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching renderImage!\n", cudaStatus);
		goto Error;
	}
	subend_t = clock();
	printf("hipDeviceSynchronize has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

	//Make random table.

	cudaStatus = hipMalloc((void**)& dev_importance, RTGRIDNUM * RTGRIDNUM * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& dev_RotationMatrix, 16 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_RotationMatrix, RotationMatrix, 16 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	estImportance <<<rtblocks, rtthreads >>> (dev_importance, g, iteration, dev_RotationMatrix);
	subend_t = clock();
	printf("Esting importance has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "estImportance launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching renderImage!\n", cudaStatus);
		goto Error;
	}
	subend_t = clock();
	printf("hipDeviceSynchronize has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

	int* importance = (int*)malloc(RTGRIDNUM * RTGRIDNUM * sizeof(int));
	for (int i = 0; i < RTGRIDNUM * RTGRIDNUM; i++) {
		importance[i] = 0;
	}

	cudaStatus = hipMemcpy(importance, dev_importance, sizeof(int) * RTGRIDNUM * RTGRIDNUM, hipMemcpyDeviceToHost);
	hipFree(dev_importance);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}
	subend_t = clock();
	printf("hipMemcpy dev_importance to importance has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

	int sum = 0, rtindex = 0;
	complex c;

	for (int i = 0; i < RTGRIDNUM; i++) {
		for (int j = 0; j < RTGRIDNUM; j++) {
			if (checkImportance(importance, i, j))
				sum++;
		}
	}
	subend_t = clock();
	printf("Computing randTable length has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

	complex* randTable = (complex*)malloc(sizeof(complex) * sum);

	for (int i = 0; i < RTGRIDNUM; i++) {
		for (int j = 0; j < RTGRIDNUM; j++) {
			if (checkImportance(importance, i, j)) {
				c.real = -3.2f + 6.4f * i / RTGRIDNUM;
				c.imag = -3.2f + 6.4f * j / RTGRIDNUM;
				randTable[rtindex] = c;
				rtindex++;
			}
		}
	}
	subend_t = clock();
	printf("Makin random table has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

	free(importance);

	// Allocate GPU buffers for a vectors (one output).
	cudaStatus = hipMalloc((void**)& dev_buddha, g.w * g.h * sizeof(unsigned long long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)& dev_randTable, sum * sizeof(complex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_buddha, buddha, g.w * g.h * sizeof(unsigned long long int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_randTable, randTable, sum * sizeof(complex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	// Compute buddhabrot.
	computeBuddhabrot <<<blocks, threads>>> (dev_buddha, g, iteration, dev_RotationMatrix, dev_states, dev_randTable, sum);
	subend_t = clock();
	printf("Computing buddhabrot has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeBuddhabrot launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching renderImage!\n", cudaStatus);
		goto Error;
	}
	subend_t = clock();
	printf("hipDeviceSynchronize has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);
	
	//Copy output vectors from GPU buffers to host memory.
	cudaStatus = hipMemcpy(buddha, dev_buddha, g.w * g.h * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}
	subend_t = clock();
	printf("cudaMencpy from dev_buddha to buddha has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

Error:
	hipFree(dev_states);
	hipFree(dev_buddha);
	hipFree(dev_randTable);

	free(randTable);

	return cudaStatus;
}

void matrix_product(float* M, const float* N) {
	float result[16] = { 0 };

	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			for (int n = 0; n < 4; n++) {
				result[j + 4 * i] += M[n + 4 * i] * N[j + 4 * n];
			}
		}
	}

	for (int i = 0; i < 16; i++) {
		M[i] = result[i];
	}
}

void set_param(int argc, char** argv) {
	int tmp;

	// Subsitute to parameters.
	if (argc > 1) {
		for (int i = 1; i < argc;) {
			if (strcmp(argv[i], "-w") == 0) {
				g.w = strtol(argv[i], NULL, 10);
				
			}
			else if (strcmp(argv[i], "-h") == 0) {
				g.h = strtol(argv[++i], NULL, 10);
			}
			else if (strcmp(argv[i], "-c") == 0) {
				g.center.real = strtof(argv[++i], NULL);
				g.center.imag = strtof(argv[++i], NULL);
			}
			else if (strcmp(argv[i], "-s") == 0) {
				g.size = strtof(argv[++i], NULL);
			}
			else if (strcmp(argv[i], "-g") == 0) {
				g.gamma = strtod(argv[++i], NULL);
			}
			else if (strcmp(argv[i], "-max") == 0) {
				iteration.max_iteration = strtol(argv[++i], NULL, 10);
			}
			else if (strcmp(argv[i], "-min") == 0) {
				iteration.min_iteration = strtol(argv[++i], NULL, 10);
			}
			else if (strcmp(argv[i], "-sample") == 0) {
				iteration.samples_per_thread = strtol(argv[++i], NULL, 10);
			}
			else if (strcmp(argv[i], "-r") == 0) {
				tmp = strtol(argv[++i], NULL, 10);
				rotation[tmp].angl = 180 * strtof(argv[++i], NULL) / 3.14159265359f;
			}
			else {
				fprintf(stderr, "Invalid options !");
				exit(1);
			}
			i++;
		}
	}

	// Compute
	g.ratio = ((double)g.w) / g.h;
	g.dx = g.size / g.h;
	g.dy = g.size / g.h;
	g.max_real = g.center.real + 0.5f * g.size * g.ratio;
	g.max_imag = g.center.imag + 0.5f * g.size;
	g.min_real = g.center.real - 0.5f * g.size * g.ratio;
	g.min_imag = g.center.imag - 0.5f * g.size;

	for (int n = 0; n < 6; n++) {
		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				rotation[n].RotMat[j + 4 * i] = (i == j) ? 1.0f : 0.0f;
			}
		}
		rotation[n].RotMat[rotation[n].axi1 + 4 * rotation[n].axi1] = cospif(rotation[n].angl);
		rotation[n].RotMat[rotation[n].axi2 + 4 * rotation[n].axi1] = -sinpif(rotation[n].angl);
		rotation[n].RotMat[rotation[n].axi1 + 4 * rotation[n].axi2] = sinpif(rotation[n].angl);
		rotation[n].RotMat[rotation[n].axi2 + 4 * rotation[n].axi2] = cospif(rotation[n].angl);
	}
	for (int i = 0; i < 16; i++) {
			RotationMatrix[i] = rotation[0].RotMat[i];
	}
	for (int n = 1; n < 6; n++) {
		matrix_product(RotationMatrix, rotation[n].RotMat);
	}
}

int main(int argc, char** argv)
{
	start_t = clock();

	// Default value.
	g.w = WIDTH;
	g.h = HEIGHT;
	g.center.real = -0.5f; // -0.15943359375f;
	g.center.imag = 0.0f; // 1.034150390625f;
	g.size = 2.6f;// 0.03125f;
	g.gamma = 1.0;

	iteration.samples_per_thread = 2;
	iteration.min_iteration = 0;
	iteration.max_iteration = 1000;

	float angles[6] = { 45.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
	for (int i=0; i < 6; i++){
		rotation[i].axi1 = rotation_axis[2 * i];
		printf("rotation axis1: %d\n", rotation_axis[2 * i]);
		rotation[i].axi2 = rotation_axis[2 * i + 1];
		printf("rotation axis2: %d\n", rotation_axis[2 * i + 1]);
		rotation[i].angl = -angles[i] / 180;
		printf("rotation angle: %f\n", rotation[i].angl);
	}

	set_param(argc, argv);

	unsigned long long int* buddha = (unsigned long long int*)malloc(sizeof(unsigned long long int) * g.w * g.h);
	if (buddha == NULL) {
		printf("Memory cannot be allocated.\n");
		free(buddha);
		return 1;
	}
	for (int i = 0; i < g.w * g.h; i++) {
		buddha[i] = 0;
	}

	// compute and render buddhabrot.
	hipError_t cudaStatus = renderImage(buddha);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "renderImage failed!\n");
		return 1;
	}

	// save image of buddhabrot.
	saveImage(buddha);
	subend_t = clock();
	printf("Saving image has done. (%.2f)\n", (double)(subend_t - start_t) / CLOCKS_PER_SEC);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	free(buddha);

    return 0;
}
